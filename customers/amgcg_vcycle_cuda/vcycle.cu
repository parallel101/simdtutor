#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#if __GNUC__ && __linux__
#include <sys/ptrace.h>

[[noreturn]] static void cuerr() {
    if (ptrace(PTRACE_TRACEME, 0, NULL, NULL) != 0)
        __builtin_trap();
    exit(EXIT_FAILURE);
}
#elif _WIN32 && _MSC_VER
#include <windows.h>

[[noreturn]] static void cuerr() {
    int debugger_present = 0;
    HANDLE process = GetCurrentProcess();
    CheckRemoteDebuggerPresent(process, &debugger_present);
    if (debugger_present) {
        __debugbreak();
    }
    exit(EXIT_FAILURE);
}
#else
[[noreturn]] static void cuerr() {
    exit(EXIT_FAILURE);
}
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipGetErrorString(status), status, #func);                     \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               hipsparseGetErrorString(status), status, #func);                 \
        cuerr();                                                               \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("%s:%d: %s (%d): %s\n", __FILE__, __LINE__,                     \
               cublasGetStatusString(status), status, #func);                  \
        cuerr();                                                               \
    }                                                                          \
}

namespace {

struct Buffer {
    void *m_data;
    size_t m_cap;

    Buffer() noexcept : m_data(nullptr), m_cap(0) {
    }

    Buffer(Buffer &&that) noexcept : m_data(that.m_data), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_cap = 0;
    }

    Buffer &operator=(Buffer &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_cap = 0;
        return *this;
    }

    ~Buffer() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void reserve(size_t new_cap) {
        if (m_cap < new_cap) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, new_cap));
            m_cap = new_cap;
        }
    }

    size_t capacity() const noexcept {
        return m_cap;
    }

    void const *data() const noexcept {
        return m_data;
    }

    void *data() noexcept {
        return m_data;
    }
};

template <class T>
hipDataType cudaDataTypeFor();

template <>
hipDataType cudaDataTypeFor<int8_t>() {
    return HIP_R_8I;
}

template <>
hipDataType cudaDataTypeFor<uint8_t>() {
    return HIP_R_8U;
}

template <>
hipDataType cudaDataTypeFor<int16_t>() {
    return HIP_R_16I;
}

template <>
hipDataType cudaDataTypeFor<uint16_t>() {
    return HIP_R_16U;
}

template <>
hipDataType cudaDataTypeFor<int32_t>() {
    return HIP_R_32I;
}

template <>
hipDataType cudaDataTypeFor<uint32_t>() {
    return HIP_R_32U;
}

template <>
hipDataType cudaDataTypeFor<int64_t>() {
    return HIP_R_64I;
}

template <>
hipDataType cudaDataTypeFor<uint64_t>() {
    return HIP_R_64U;
}

template <>
hipDataType cudaDataTypeFor<nv_half>() {
    return HIP_R_16F;
}

template <>
hipDataType cudaDataTypeFor<hip_bfloat16>() {
    return HIP_R_16BF;
}

template <>
hipDataType cudaDataTypeFor<float>() {
    return HIP_R_32F;
}

template <>
hipDataType cudaDataTypeFor<double>() {
    return HIP_R_64F;
}

template <class T>
struct Vec {
    T *m_data;
    size_t m_size;
    size_t m_cap;

    Vec() noexcept : m_data(nullptr), m_size(0), m_cap(0) {
    }

    Vec(Vec &&that) noexcept : m_data(that.m_data), m_size(that.m_size), m_cap(that.m_cap) {
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
    }

    Vec &operator=(Vec &&that) noexcept {
        if (this == &that) return *this;
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
        m_data = that.m_data;
        m_size = that.m_size;
        m_cap = that.m_cap;
        that.m_data = nullptr;
        that.m_size = 0;
        that.m_cap = 0;
        return *this;
    }

    void swap(Vec &that) noexcept {
        std::swap(m_data, that.m_data);
        std::swap(m_size, that.m_size);
        std::swap(m_cap, that.m_cap);
    }

    ~Vec() noexcept {
        if (m_data)
            CHECK_CUDA(hipFree(m_data));
        m_data = nullptr;
    }

    void resize(size_t new_size) {
        bool change = m_cap < new_size;
        if (change) {
            if (m_data)
                CHECK_CUDA(hipFree(m_data));
            m_data = nullptr;
            CHECK_CUDA(hipMalloc(&m_data, sizeof(T) * new_size));
            m_cap = new_size;
        }
        if (m_size != new_size || change) {
            m_size = new_size;
        }
    }

    void assign(T const *data, size_t size) {
        resize(size);
        CHECK_CUDA(hipMemcpy(m_data, data, sizeof(T) * size, hipMemcpyHostToDevice));
    }

    void store(T *data) const {
        CHECK_CUDA(hipMemcpy(data, m_data, sizeof(T) * size(), hipMemcpyDeviceToHost));
    }

    size_t size() const noexcept {
        return m_size;
    }

    T const *data() const noexcept {
        return m_data;
    }

    T *data() noexcept {
        return m_data;
    }
};

struct DnVec {
    hipsparseDnVecDescr_t handle;

    operator hipsparseDnVecDescr_t() const noexcept {
        return handle;
    }

    DnVec() noexcept : handle(0) {}

    template <class T>
    DnVec(Vec<T> &v) {
        CHECK_CUSPARSE(hipsparseCreateDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    DnVec(DnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    DnVec &operator=(DnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~DnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

struct ConstDnVec {
    hipsparseConstDnVecDescr_t handle;

    operator hipsparseConstDnVecDescr_t() const noexcept {
        return handle;
    }

    ConstDnVec() noexcept : handle(0) {}

    template <class T>
    ConstDnVec(Vec<T> const &v) {
        CHECK_CUSPARSE(hipsparseCreateConstDnVec(&handle, v.size(), v.data(), cudaDataTypeFor<T>()));
    }

    ConstDnVec(ConstDnVec &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstDnVec &operator=(ConstDnVec &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstDnVec() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroyDnVec(handle));
    }
};

template <class T>
struct CSR {
    Vec<T> data;
    Vec<int> indices;
    Vec<int> indptr;
    size_t nrows;
    size_t ncols;
    size_t numnonz;

    CSR() noexcept : nrows(0), ncols(0), numnonz(0) {}

    void assign(T const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        data.resize(ndat);
        CHECK_CUDA(hipMemcpy(data.data(), datap, data.size() * sizeof(T), hipMemcpyHostToDevice));
        indices.resize(nind);
        CHECK_CUDA(hipMemcpy(indices.data(), indicesp, indices.size() * sizeof(int), hipMemcpyHostToDevice));
        indptr.resize(nptr);
        CHECK_CUDA(hipMemcpy(indptr.data(), indptrp, indptr.size() * sizeof(int), hipMemcpyHostToDevice));
        nrows = rows;
        ncols = cols;
        numnonz = nnz;
    }
};

struct SpMat {
    hipsparseSpMatDescr_t handle;

    operator hipsparseSpMatDescr_t() const noexcept {
        return handle;
    }

    SpMat() noexcept : handle(0) {}

    template <class T>
    SpMat(CSR<T> &m) {
        CHECK_CUSPARSE(hipsparseCreateCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                         m.indptr.data(), m.indices.data(), m.data.data(),
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    SpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    SpMat &operator=(SpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~SpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct ConstSpMat {
    hipsparseConstSpMatDescr_t handle;

    operator hipsparseConstSpMatDescr_t() const noexcept {
        return handle;
    }

    ConstSpMat() noexcept : handle(0) {}

    template <class T>
    ConstSpMat(CSR<T> const &m) {
        CHECK_CUSPARSE(hipsparseCreateConstCsr(&handle, m.nrows, m.ncols, m.numnonz,
                                              m.indptr.data(), m.indices.data(), m.data.data(),
                                              HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                              HIPSPARSE_INDEX_BASE_ZERO, cudaDataTypeFor<T>()) );
    }

    ConstSpMat(SpMat &&that) noexcept : handle(that.handle) {
        that.handle = nullptr;
    }

    ConstSpMat &operator=(ConstSpMat &&that) noexcept {
        if (this == &that) return *this;
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
        handle = that.handle;
        that.handle = nullptr;
        return *this;
    }

    ~ConstSpMat() {
        if (handle)
            CHECK_CUSPARSE(hipsparseDestroySpMat(handle));
    }
};

struct Kernels {
    hipblasHandle_t cublas;
    hipsparseHandle_t cusparse;

    Kernels() {
        CHECK_CUSPARSE(hipsparseCreate(&cusparse));
        CHECK_CUBLAS(hipblasCreate(&cublas));
    }

    Kernels(Kernels &&) = delete;

    ~Kernels() {
        CHECK_CUSPARSE(hipsparseDestroy(cusparse));
        CHECK_CUBLAS(hipblasDestroy(cublas));
    }

    // out = alpha * A@x + beta * out
    void spmv(Vec<float> &out, float const &alpha, CSR<float> const &A, Vec<float> const &x, float const &beta, Buffer &buffer) {
        assert(out.size() == A.nrows);
        size_t bufSize = 0;
        ConstSpMat dA(A);
        ConstDnVec dx(x);
        DnVec dout(out);
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha, dA, dx, &beta,
                                               dout, cudaDataTypeFor<float>(),
                                               HIPSPARSE_SPMV_ALG_DEFAULT, &bufSize));
        buffer.reserve(bufSize);
        CHECK_CUSPARSE(hipsparseSpMV(cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, dA, dx, &beta,
                                    dout, cudaDataTypeFor<float>(),
                                    HIPSPARSE_SPMV_ALG_DEFAULT, buffer.data()));
    }

    // dst = src + alpha * dst
    void axpy(Vec<float> &dst, float const &alpha, Vec<float> const &src) {
        assert(dst.size() == src.size());
        CHECK_CUBLAS(hipblasSaxpy(cublas, dst.size(), &alpha, src.data(), 1, dst.data(), 1));
    }

    void zero(Vec<float> &dst) {
        CHECK_CUDA(hipMemset(dst.data(), 0, dst.size() * sizeof(float)));
    }

    void copy(Vec<float> &dst, Vec<float> const &src) {
        dst.resize(src.size());
        CHECK_CUDA(hipMemcpy(dst.data(), src.data(), src.size() * sizeof(float), hipMemcpyDeviceToDevice));
    }

    // dst = alpha * x
    void scal2(Vec<float> &dst, float const &alpha, Vec<float> const &x) {
        copy(dst, x);
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // dst = alpha * alpha
    void scal(Vec<float> &dst, float const &alpha) {
        CHECK_CUBLAS(hipblasSscal(cublas, dst.size(), &alpha, dst.data(), 1));
    }

    // x = A^{-1} b
    void spsolve(Vec<float> &x, CSR<float> const &A, Vec<float> const &b) {
        throw;
    }

    float vdot(Vec<float> const &x, Vec<float> const &y) {
        float result;
        CHECK_CUBLAS(hipblasSdot(cublas, x.size(), x.data(), 1, y.data(), 1, &result));
        return result;
    }

    float vnorm(Vec<float> const &x) {
        float result;
        CHECK_CUBLAS(hipblasSnrm2(cublas, x.size(), x.data(), 1, &result));
        return result;
    }
};

struct MGLevel {
    CSR<float> A;
    CSR<float> R;
    CSR<float> P;
    Vec<float> residual;
    Vec<float> b;
    Vec<float> x;
    Vec<float> h;
    Vec<float> outh;
};

struct VCycle : Kernels {
    std::vector<MGLevel> levels;
    size_t nlvs;
    std::vector<float> coefficients;
    Vec<float> init_x;
    Vec<float> init_b;
    Vec<float> outer_x;
    Vec<float> alter_x;
    Vec<float> outer_b;
    float save_rho_prev;
    Vec<float> save_p;
    Vec<float> save_q;
    Buffer buff;

    void setup(size_t numlvs) {
        if (levels.size() < numlvs) {
            levels.resize(numlvs);
        }
        nlvs = numlvs;
        coefficients.clear();
    }

    void set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
        CSR<float> *mat = nullptr;
        if (which == 1) mat = &levels.at(lv).A;
        if (which == 2) mat = &levels.at(lv).R;
        if (which == 3) mat = &levels.at(lv).P;
        if (mat) {
            mat->assign(datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
        }
    }

    void set_coeff(float const *coeff, size_t ncoeffs) {
        coefficients.assign(coeff, coeff + ncoeffs);
    }

    void _smooth(int lv, Vec<float> &x, Vec<float> const &b) {
        copy(levels.at(lv).residual, b);
        spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x
        scal2(levels.at(lv).h, coefficients.at(0), levels.at(lv).residual); // h = c0 * residual


        for (int i = 1; i < coefficients.size(); ++i) {
            // h' = ci * residual + A@h
            copy(levels.at(lv).outh, levels.at(lv).residual);
            spmv(levels.at(lv).outh, 1, levels.at(lv).A, levels.at(lv).h, coefficients.at(i), buff);

            // copy(levels.at(lv).h, levels.at(lv).outh);
            levels.at(lv).h.swap(levels.at(lv).outh);
        }

        axpy(x, 1, levels.at(lv).h); // x += h
    }

    void set_init_x(float const *x, size_t n) {
        init_x.resize(n);
        CHECK_CUDA(hipMemcpy(init_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
    }

    void set_init_b(float const *b, size_t n) {
        init_b.resize(n);
        CHECK_CUDA(hipMemcpy(init_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    void vcycle_down() {
        for (int lv = 0; lv < nlvs-1; ++lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            _smooth(lv, x, b);

            copy(levels.at(lv).residual, b);
            spmv(levels.at(lv).residual, -1, levels.at(lv).A, x, 1, buff); // residual = b - A@x

            levels.at(lv).b.resize(levels.at(lv).R.nrows);
            spmv(levels.at(lv).b, 1, levels.at(lv).R, levels.at(lv).residual, 0, buff); // coarse_b = R@residual

            levels.at(lv).x.resize(levels.at(lv).b.size());
            zero(levels.at(lv).x);
        }
    }

    void vcycle_up() {
        for (int lv = nlvs-2; lv >= 0; --lv) {
            Vec<float> &x = lv != 0 ? levels.at(lv - 1).x : init_x;
            Vec<float> &b = lv != 0 ? levels.at(lv - 1).b : init_b;
            spmv(x, 1, levels.at(lv).P, levels.at(lv).x, 1, buff); // x += P@coarse_x
            _smooth(lv, x, b);
        }
    }

    size_t get_coarsist_size() {
        auto const &this_b = levels.at(nlvs - 2).b;
        return this_b.size();
    }

    void get_coarsist_b(float *b) {
        auto const &this_b = levels.at(nlvs - 2).b;
        CHECK_CUDA(hipMemcpy(b, this_b.data(), this_b.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void get_finest_x(float *x) {
        CHECK_CUDA(hipMemcpy(x, init_x.data(), init_x.size() * sizeof(float), hipMemcpyDeviceToHost));
    }

    void set_coarsist_x(float const *x) {
        auto const &this_b = levels.at(nlvs - 2).b;
        auto &this_x = levels.at(nlvs - 2).x;
        this_x.resize(this_b.size());
        CHECK_CUDA(hipMemcpy(this_x.data(), x, this_x.size() * sizeof(float), hipMemcpyHostToDevice));
    }

    void coarse_solve() {
        auto const &A = levels.at(nlvs - 1).A;
        auto &x = levels.at(nlvs - 2).x;
        auto const &b = levels.at(nlvs - 2).b;
        spsolve(x, A, b);
    }

    void copy_outer2init_x() {
        copy(init_x, outer_x);
    }

    void set_outer_x(float const *x, size_t n) {
        outer_x.resize(n);
        CHECK_CUDA(hipMemcpy(outer_x.data(), x, n * sizeof(float), hipMemcpyHostToDevice));
        copy(alter_x, outer_x);
    }

    void set_outer_b(float const *b, size_t n) {
        outer_b.resize(n);
        CHECK_CUDA(hipMemcpy(outer_b.data(), b, n * sizeof(float), hipMemcpyHostToDevice));
    }

    float init_cg_iter0(float *residuals) {
        float bnrm2 = vnorm(outer_b);
        // r = b - A@(x)
        copy(init_b, outer_b);
        spmv(outer_b, -1, levels.at(0).A, outer_x, 1, buff);
        float normr = vnorm(init_b);
        residuals[0] = normr;
        return bnrm2;
    }

    void do_cg_itern(float *residuals, size_t iteration) {
        float rho_cur = vdot(init_b, init_x);
        if (iteration > 0) {
            float beta = rho_cur / save_rho_prev;
            // p *= beta
            // p += z
            scal(save_p, beta);
            axpy(save_p, 1, init_x);
        } else {
            // p = move(z)
            save_p.swap(init_x);
        }
        // q = A@(p)
        save_q.resize(levels.at(0).A.nrows);
        spmv(save_q, 1, levels.at(0).A, save_p, 0, buff);
        save_rho_prev = rho_cur;
        float alpha = rho_cur / vdot(save_p, save_q);
        // x += alpha*p
        axpy(alter_x, alpha, save_p);
        // r -= alpha*q
        axpy(init_b, -alpha, save_q);
        float normr = vnorm(init_b);
        residuals[iteration + 1] = normr;
    }

    void fetch_cg_final_x(float *x) {
        CHECK_CUDA(hipMemcpy(x, alter_x.data(), alter_x.size() * sizeof(float), hipMemcpyDeviceToHost));
    }
};

}

static VCycle *fastmg = nullptr;

#if _WIN32
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif

extern "C" DLLEXPORT void fastmg_setup(size_t numlvs) {
    if (!fastmg)
        fastmg = new VCycle{};
    fastmg->setup(numlvs);
}

extern "C" DLLEXPORT void fastmg_set_coeff(float const *coeff, size_t ncoeffs) {
    fastmg->set_coeff(coeff, ncoeffs);
}

extern "C" DLLEXPORT void fastmg_set_lv_csrmat(size_t lv, size_t which, float const *datap, size_t ndat, int const *indicesp, size_t nind, int const *indptrp, size_t nptr, size_t rows, size_t cols, size_t nnz) {
    fastmg->set_lv_csrmat(lv, which, datap, ndat, indicesp, nind, indptrp, nptr, rows, cols, nnz);
}

extern "C" DLLEXPORT void fastmg_set_init_x(float const *x, size_t n) {
    fastmg->set_init_x(x, n);
}

extern "C" DLLEXPORT void fastmg_set_init_b(float const *b, size_t n) {
    fastmg->set_init_b(b, n);
}

extern "C" DLLEXPORT void fastmg_vcycle_down() {
    fastmg->vcycle_down();
}

extern "C" DLLEXPORT void fastmg_vcycle_up() {
    fastmg->vcycle_up();
}

extern "C" DLLEXPORT size_t fastmg_get_coarsist_size() {
    return fastmg->get_coarsist_size();
}

extern "C" DLLEXPORT void fastmg_get_coarsist_b(float *b) {
    fastmg->get_coarsist_b(b);
}

extern "C" DLLEXPORT void fastmg_set_coarsist_x(float const *x) {
    fastmg->set_coarsist_x(x);
}

extern "C" DLLEXPORT void fastmg_get_finest_x(float *x) {
    fastmg->get_finest_x(x);
}

extern "C" DLLEXPORT void fastmg_coarse_solve() {
    fastmg->coarse_solve();
}

extern "C" DLLEXPORT void fastmg_set_outer_x(float const *x, size_t n) {
    fastmg->set_outer_x(x, n);
}

extern "C" DLLEXPORT void fastmg_copy_outer2init_x() {
    fastmg->copy_outer2init_x();
}

extern "C" DLLEXPORT void fastmg_set_outer_b(float const *b, size_t n) {
    fastmg->set_outer_b(b, n);
}

extern "C" DLLEXPORT float fastmg_init_cg_iter0(float *residuals) {
    return fastmg->init_cg_iter0(residuals);
}

extern "C" DLLEXPORT void fastmg_do_cg_itern(float *residuals, size_t iteration) {
    return fastmg->do_cg_itern(residuals, iteration);
}

extern "C" DLLEXPORT void fastmg_fetch_cg_final_x(float *x) {
    fastmg->fetch_cg_final_x(x);
}
