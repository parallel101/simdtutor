#include <omp.h>
#include <chrono>
#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"


#define checkCudaErrors(x) do { hipError_t __ret = (x); if (__ret) { printf("CUDA ERROR %d: " #x "\n", __ret); abort(); } } while (0)

#define TYPE double
#define imgW 2448
#define imgH 2048
#define N (imgW*imgH)

__constant__ TYPE c_para0[] = {1.5, 1.5, 1.5, 1.5, 1.5, 1.5};
__constant__ TYPE c_para1[] = {1.5, 1.5, 1.5, 1.5, 1.5, 1.5};
__constant__ TYPE c_para2[] = {1246, 1037, 2448, 2048};

#if 1
__global__ void GPU_Cal(TYPE *input, TYPE *output, int width, int height, TYPE *para0, TYPE *para1,
                                      TYPE *para2) {
    // 2d grid stride loop
    for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < height; row += blockDim.y * gridDim.y) {
        for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < width; col += blockDim.x * gridDim.x) {
            int i = row * width + col;
            TYPE data = input[i];
            TYPE x = (row - para2[0]) * para2[2];
            TYPE y = (col - para2[1]) * para2[3];

            const TYPE a = para0[0] + para0[2] * x + data * (para0[1] + para0[3] * x) + para0[4] * y + data * para0[5] * y;
            const TYPE b = para1[0] + para1[2] * x + data * (para1[1] + para1[3] * x) + para1[4] * y + data * para1[5] * y;

            output[i] = a / b;
        }
    }
}
#else
__global__ void GPU_Cal(TYPE *input, TYPE *output, int width, int height, TYPE *para0, TYPE *para1,
                                      TYPE *para2) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < width * height; i += gridDim.x * blockDim.x) {
        TYPE data = input[i];
        int row = i / width;
        int col = i % height;
        TYPE x = (col - para2[0]) * para2[2];
        TYPE y = (row - para2[1]) * para2[3];

        const TYPE a = para0[0] + para0[2] * x + data * (para0[1] + para0[3] * x) + para0[4] * y + data * para0[5] * y;
        const TYPE b = para1[0] + para1[2] * x + data * (para1[1] + para1[3] * x) + para1[4] * y + data * para1[5] * y;

        output[i] = a / b;
    }
}
#endif

void CPU_Cal(const TYPE *input, TYPE *output, int width, int height, TYPE *para0, TYPE *para1, TYPE *para2) {
#pragma omp parallel for
    for (int row = 0; row < height; ++row) {
        TYPE *_output = output + row * width;
        const TYPE *_input = input + row * width;
        for (int col = 0; col < width; ++col) {
            const TYPE data = *_input;
            const TYPE x = (col - para2[0]) * para2[2];
            const TYPE y = (row - para2[1]) * para2[3];

            const TYPE a =
                para0[0] + para0[2] * x + data * (para0[1] + para0[3] * x) + para0[4] * y + data * para0[5] * y;
            const TYPE b =
                para1[0] + para1[2] * x + data * (para1[1] + para1[3] * x) + para1[4] * y + data * para1[5] * y;

            *_output = a / b; 
            ++_output;
            ++_input;
        }
    }
}

int main() {
    // 准备数据
    std::vector<TYPE> input(N, 2);
    std::vector<TYPE> output(N, 0);
    std::vector<TYPE> para0(30, 1.5);
    std::vector<TYPE> para1(30, 1.5);
    std::vector<TYPE> para3{1246, 1037, 2448, 2048};
    // 随机准备一段数据
    for (int i = 0; i < N; ++i) {
        input[i] = (double)i / N;
        output[i] = (double)i / N + 2;
    }
    for (int i = 0; i < 30; ++i) {
        para0[i] = (double)i / 30;
        para1[i] = (double)i / 30 + 4.0;
    }

    TYPE *d_input;
    TYPE *d_output;
    TYPE *d_para0;
    TYPE *d_para1;
    TYPE *d_para2;
    hipMalloc((void **)&d_input, N * sizeof(TYPE));
    hipMalloc((void **)&d_output, N * sizeof(TYPE));
    hipMalloc((void **)&d_para0, 30 * sizeof(TYPE));
    hipMalloc((void **)&d_para1, 30 * sizeof(TYPE));
    hipMalloc((void **)&d_para2, 4 * sizeof(TYPE));
    hipMemcpy(d_input, input.data(), N * sizeof(TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output.data(), N * sizeof(TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_para0, para0.data(), 30 * sizeof(TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_para1, para1.data(), 30 * sizeof(TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_para2, para3.data(), 4 * sizeof(TYPE), hipMemcpyHostToDevice);

    // GPU计算时间（取最短时间）
    dim3 thread_num = dim3(32, 32, 1);
    dim3 block_num = dim3(256, 256, 1);
    double gpu_time = 10000000;
    checkCudaErrors(hipDeviceSynchronize());
    for (size_t i = 0; i < 50; i++) {
        auto t0 = std::chrono::steady_clock::now();
        GPU_Cal<<<block_num, thread_num>>>(d_input, d_output, imgW, imgH, d_para0, d_para1, d_para2);
        checkCudaErrors(hipDeviceSynchronize());
        double time =
            std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - t0).count();
        gpu_time = std::min(gpu_time, time);
    }
    std::cout << "GPU time: " << gpu_time << std::endl;

    // CPU计算时间（取最短时间）
    TYPE *h_output;
    h_output = (TYPE *)malloc(N * sizeof(TYPE));
    hipMemcpy(h_output, d_output, N * sizeof(TYPE), hipMemcpyDeviceToHost);
    double cpu_time = 10000000;
    for (size_t i = 0; i < 50; i++) {
        auto t0 = std::chrono::steady_clock::now();
        CPU_Cal(input.data(), output.data(), imgW, imgH, para0.data(), para1.data(), para3.data());
        double time =
            std::chrono::duration_cast<std::chrono::duration<double>>(std::chrono::steady_clock::now() - t0).count();
        cpu_time = std::min(cpu_time, time);
    }
    std::cout << "CPU time: " << cpu_time << std::endl;
    std::cout << "ratio: " << cpu_time / gpu_time << std::endl;

    // 检测计算结果是否一致
    for (int i = 0; i < N; i++) {
        if (h_output[i] != h_output[i] && output[i] != output[i]) {
            continue;
        }
        if (fabs(h_output[i] - output[i]) > 1e-2) {
            printf("Error! i: %d, cpu: %f, gpu:%f.\n", i, output[i], h_output[i]);
            abort();
        }
    }

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_para0);
    hipFree(d_para1);
    hipFree(d_para2);
    return 0;
}
